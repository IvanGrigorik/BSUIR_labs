
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <chrono>

using namespace std::chrono;

using namespace std;

#define N /*16*/ /*32*/ /*64*/ 128

__managed__
int matrix[N][N];

void fill_matrix() {
    unsigned n = 1;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i][j] = n++;
        }
    }
}

__managed__ int out_matrix_gpu[N][N];
int out_matrix_cpu[N][N];

__global__ void mash_gpu() {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned i = idx / N % N;
    unsigned j = idx % N;
    auto i_1 = i, j_1 = j;

    auto quarter_i = i % 2;
    auto quarter_j = j % 2;

    quarter_i == 0 ? i /= 2 : i += (N - i) / 2;
    quarter_j == 0 ? j /= 2 : j += (N - j) / 2;

    out_matrix_gpu[i][j] = matrix[i_1][j_1];
}

void mash_cpu() {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            auto i_out = i;
            auto j_out = j;

            auto quarter_i = i % 2;
            auto quarter_j = j % 2;

            if (quarter_i == 0) {
                i_out /= 2;
            } else if (quarter_i == 1) {
                i_out = i_out + (N - i_out) / 2;
            }

            if (quarter_j == 0) {
                j_out /= 2;
            } else if (quarter_j == 1) {
                j_out = j_out + (N - j_out) / 2;
            }

            out_matrix_cpu[i_out][j_out] = matrix[i][j];
        }
    }
};

void run_gpu() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    mash_gpu<<<N + 127 / 128, 128>>>();
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    cout << "GPU elapsed time: " << milliseconds << endl;
    hipDeviceSynchronize();

}

void run_cpu() {

    auto start = steady_clock::now();
    mash_cpu();
    auto time = duration_cast<microseconds>(
            steady_clock::now() - start).count();
    cout << "CPU elapsed time: " << static_cast<double>(time) / 1000 << endl;
}

bool is_equal() {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (out_matrix_cpu[i][j] != out_matrix_gpu[i][j]) {
                return false;
            }
        }
    }
    return true;
}

int main() {

    fill_matrix();

    run_gpu();
    run_cpu();

    cout << "Is matrix equal: " << std::boolalpha << is_equal() << endl;
}


//void matrix_output() {
//    cout << "Initial matrix: " << endl;
//    for (int i = 0; i < N; i++) {
//        cout << "|";
//        for (int j = 0; j < N; j++) {
//            cout << setw(3) << matrix[i][j] << ' ';
//            if ((j + 1) % 2 == 0) {
//                cout << "| ";
//            }
//        }
//        if ((i + 1) % 2 == 0 && i + 1 != N) {
//            cout << endl;
//            for (int k = 0; k < N; k++) {
//                cout << setw(3) << "-----";
//            }
//        }
//        cout << endl;
//    }
//
//    cout << endl << "Result matrix: " << endl;
//    for (int i = 0; i < N; i++) {
//        for (int j = 0; j < N; j++) {
//            cout << setw(3) << out_matrix[i][j] << ' ';
//            if (j == N / 2 - 1) {
//                cout << setw(3) << "| ";
//            }
//        }
//        if (i == N / 2 - 1) {
//            cout << endl;
//            for (int k = 0; k < N; k++) {
//                cout << setw(3) << "----";
//            }
//        }
//
//        cout << endl;
//    }
//}