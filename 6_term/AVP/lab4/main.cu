#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <hiprand/hiprand_kernel.h>
#include <chrono>

#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

using namespace std;
using namespace chrono;

typedef struct Point {
    float x{};
    float y{};
    float z{};

    Point() = default;

    Point(double a_x, double a_y, double a_z);

    Point &operator=(const Point &a) = default;

} Point;

Point::Point(double a_x, double a_y, double a_z) {
    x = static_cast<float>(a_x);
    y = static_cast<float>(a_y);
    z = static_cast<float>(a_z);
}

typedef struct Plane {
    float x;
    float y;
    float z;
    float d;

    Plane &operator=(const Plane &a) = default;
} Plane;

Plane define_plane(Point p1, Point p2, Point p3) {
    Point m1m2{}, m1m3{};
    m1m2 = {p2.x - p1.x, p2.y - p1.y, p2.z - p1.z};
    m1m3 = {p3.x - p1.x, p3.y - p1.y, p3.z - p1.z};
    Plane result_plane;

    result_plane = {m1m2.y * m1m3.z - m1m2.z * m1m3.y,
                    m1m2.z * m1m3.x - m1m2.x * m1m3.z,
                    m1m2.x * m1m3.y - m1m2.y * m1m3.x};
    result_plane.d = -p1.x * result_plane.x + -p1.y * result_plane.y +
                     -p1.z * result_plane.z;
    return result_plane;
}

__managed__
Plane planes[4]{};

const int numBlocks = 1024;

__managed__ int shared_grid[numBlocks];

__global__ void kernel(Point r_min, Point r_max, const bool *planes_signs) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState localState;
    hiprand_init(1234, idx, 0, &localState);

    Point p;
    p.x = hiprand_uniform(&localState) * (r_max.x - r_min.x) + r_min.x;
    p.y = hiprand_uniform(&localState) * (r_max.y - r_min.y) + r_min.y;
    p.z = hiprand_uniform(&localState) * (r_max.z - r_min.z) + r_min.z;

    float sides[4];
    bool signs[4]{};

    for (int j = 0; j < 4; j++) {
        sides[j] = planes[j].x * p.x +
                   planes[j].y * p.y +
                   planes[j].z * p.z +
                   planes[j].d;
        signs[j] = signbit(sides[j]);
    }

    bool is_hit = true;
    for (int i = 0; i < 4; i++) {
        if (signs[i] != planes_signs[i]) {
            is_hit = false;
            break;
        }
    }

    // Synchronize threads in block
    extern __shared__ int shared_block[];
    shared_block[threadIdx.x] = is_hit ? 1 : 0;

    __syncthreads();
    size_t stride = blockDim.x / 2;

    while (stride > 0) {
        if (threadIdx.x >= stride) {
            return;
        }

        shared_block[threadIdx.x] += shared_block[threadIdx.x + stride];
        stride /= 2;
        __syncthreads();
    }

    shared_grid[blockIdx.x] = shared_block[threadIdx.x];
}

int run_cpu(Point r_min, Point r_max, int try_counter) {
    random_device rd;
    mt19937 rng(rd());
    uniform_real_distribution<> dist_x(r_min.x, r_max.x);
    uniform_real_distribution<> dist_y(r_min.y, r_max.y);
    uniform_real_distribution<> dist_z(r_min.z, r_max.z);

    float sides[4]{};
    int hit_counter = 0;
    bool signs[4]{};
    bool planes_signs[4]{};
    for (int i = 0; i < 4; i++) {
        planes_signs[i] = signbit(planes[i].d);
    }

    for (int i = 0; i < try_counter; i++) {
        Point p{dist_x(rng), dist_y(rng), dist_z(rng)};

        for (int j = 0; j < 4; j++) {
            sides[j] = planes[j].x * p.x +
                       planes[j].y * p.y +
                       planes[j].z * p.z +
                       planes[j].d;

            signs[j] = signbit(sides[j]);
        }

        bool is_hit = true;
        for (int j = 0; j < 4; j++) {
            if (signs[j] != planes_signs[j]) {
                is_hit = false;
                break;
            }
        }

        if (is_hit) {
            hit_counter++;
        }
    }

    return hit_counter;
}


int main() {
    // GPU info output
    int nDevices{};
    hipGetDeviceCount(&nDevices);

    for (int i = 0; i < nDevices; i++) {

        hipDeviceProp_t prop{};

        hipGetDeviceProperties(&prop, 0);
        cout << "GPU: " << prop.name << endl;
        cout << "Global memory: " << prop.totalGlobalMem << endl;
        cout << "Shared memory: "
             << prop.sharedMemPerBlock * prop.sharedMemPerBlock << endl;
        cout << "Compute capability: " << prop.major << "." << prop.minor
             << endl;
    }

    Point A{0.4, 0.5, -0.05}, B{0.2, -1, -0.2},
            C{-2, -0.3, -0.5}, D{-0.1, 0.05, 0.3};

    planes[0] = define_plane(A, B, C);
    planes[1] = define_plane(A, B, D);
    planes[2] = define_plane(B, C, D);
    planes[3] = define_plane(C, A, D);

    Point points[4] = {A, B, C, D};
    Point r_min{A.x, A.y, A.z}, r_max{A.x, A.y, A.z};

    for (auto point: points) {
        r_min.x = point.x < r_min.x ? point.x : r_min.x;
        r_min.y = point.y < r_min.y ? point.y : r_min.y;
        r_min.z = point.z < r_min.z ? point.z : r_min.z;

        r_max.x = point.x > r_max.x ? point.x : r_max.x;
        r_max.y = point.y > r_max.y ? point.y : r_max.y;
        r_max.z = point.z > r_max.z ? point.z : r_max.z;
    }

//    int gpu_precision = 10'000'000;
    int cpu_precision = 51'200;
    auto main_volume =
            (r_max.x - r_min.x) * (r_max.y - r_min.y) * (r_max.z - r_min.z);

    auto start = steady_clock::now();
    auto cpu_hits = run_cpu(r_min, r_max, cpu_precision);
    auto time = duration_cast<milliseconds>(
            steady_clock::now() - start).count();
    cout << "CPU time: " << time << " milliseconds." << endl;
    cout << "CPU volume: " << static_cast<float>(cpu_hits) * main_volume /
                              static_cast<float>(cpu_precision) << endl;
    cout << "CPU hits: " << cpu_hits << endl;


    bool planes_signs[4]{};
    for (int i = 0; i < 4; i++) {
        planes_signs[i] = signbit(planes[i].d);
    }

    bool *d_planes_signs;
    hipMalloc(&d_planes_signs, 4 * sizeof(bool));
    hipMemcpy(d_planes_signs, planes_signs, 4 * sizeof(bool),
               hipMemcpyHostToDevice);

    int threadsPerBlock = 512;

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    hipEventRecord(start_event);

    kernel<<<numBlocks, threadsPerBlock, /*Shared memory size*/1024>>>(r_min, r_max, d_planes_signs);
    hipDeviceSynchronize();
    thrust::device_vector<unsigned> d_vec(shared_grid, shared_grid + numBlocks);
    auto gpu_hits = thrust::reduce(thrust::device,
                                   d_vec.begin(), d_vec.end(),
                                   0, thrust::plus<>());

    hipEventRecord(stop_event);
    hipEventSynchronize(start_event);
    hipEventSynchronize(stop_event);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start_event, stop_event);
    cout << "GPU time: " << milliseconds << endl;

    auto gpu_precision = threadsPerBlock * numBlocks;
    cout << "GPU volume: " << static_cast<float>(gpu_hits) * main_volume /
                              static_cast<float>(gpu_precision) << endl;
    cout << "GPU hits: " << gpu_hits;
    cout << gpu_hits;
    return 0;
}
