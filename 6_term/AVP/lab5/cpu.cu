#include "hip/hip_runtime.h"
#include "cpu.cuh"
#include <iostream>
#include <utility>

std::vector<std::vector<int>> houghAccum;
Pixel thresholdCPU{20, 20, 20};
Pixel markerCPU{0, 255, 0};

/// Returns line angle relative to the horizon (see explanation ahead the function)
/*
 * |                     ----
 * |                 ----
 * |             ---- A (equivalent)
 * | ****************************** horizon line
 * |       A ----
 * |     ----
 * | ----
 * |||||||||||||||||||||||||||||||| image x-axis
 * Return angle A
 */
double houghTransform(const ImageCPU &image) {
    ImageCPU monochromaticImage{"../outImages/monochromatic.png"};
    monochromaticImage.setProperties(image.getHeight(), image.getWidth(), image.getChannels());
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) {
        houghAccum[i].resize(thetasCount);
    }

    const float step = 180.f / static_cast<float>(thetasCount);

    for (int y = 0; y < image.getHeight(); y++) {
        for (int x = 0; x < image.getWidth(); x++) {
            // Find pixel fits the markerCPU

            if (abs(image.getPixel(x, y).red - markerCPU.red) >= thresholdCPU.red or      //
                abs(image.getPixel(x, y).green - markerCPU.green) >= thresholdCPU.green or//
                abs(image.getPixel(x, y).blue - markerCPU.blue) >= thresholdCPU.blue) {
                continue;
            }
            monochromaticImage.setPixel(x, y, {255, 255, 255});
            double ang = -90;
            for (int h = 0; h < thetasCount; ang += step, h++) {
                int idx = static_cast<int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
                houghAccum[idx][h]++;
            }
        }
    }


    monochromaticImage.writeImage();
    int idx{}, max{};
    for (auto &i: houghAccum) {
        for (int j = 0; j < i.size(); j++) {
            if (i[j] > max) {
                max = i[j];
                idx = j;
            }
        }
    }


    return -90 + step * static_cast<float>(idx);
}

ImageCPU rotateImage(const ImageCPU &image, const double angle) {
    ImageCPU rotatedImage{"../outImages/rotatedImage.png"};

    const auto height = image.getHeight(), width = image.getWidth();

    const int centerX = width / 2, centerY = height / 2;
    rotatedImage.setProperties(height, width, image.getChannels());

    const auto radAngle = degToRad(90 + angle);

    const auto k1 = sin(radAngle);
    const auto k2 = cos(radAngle);

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            auto newX = static_cast<int>(k1 * (x - centerX) + k2 * (y - centerY) + centerX);
            auto newY = static_cast<int>(k1 * (y - centerY) - k2 * (x - centerX) + centerY);

            if (newX < 0 or newX >= width or newY < 0 or newY >= height) {
                continue;
            }

            rotatedImage.setPixel(newX, newY, image.getPixel(x, y));
            rotatedImage.definePixel(newX, newY);
        }
    }

    rotatedImage.writeImage();
    return rotatedImage;
}

ImageCPU interpolarImage(const ImageCPU &image) {
    ImageCPU interpolaredImage{image};
    auto height = image.getHeight(), width = image.getWidth();
    ImageCPU undefinedMap{"../outImages/undefinedPixels.png"};
    undefinedMap.setProperties(height, width, image.getChannels());

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            if (!interpolaredImage.getPixel(x, y).isDefined) {
                Pixel newPixel{};

                // rotated image cannot contain more than two undefined pixels in a row
                if (x + 2 < width and !image.getPixel(x + 1, y).isDefined and !image.getPixel(x + 2, y).isDefined) {
                    continue;
                }

                int shots{};
                if (x + 1 < width and image.getPixel(x + 1, y).isDefined) {
                    newPixel = image.getPixel(x + 1, y);
                    shots++;
                }
                if (x - 1 >= 0 and image.getPixel(x - 1, y).isDefined) {
                    newPixel += image.getPixel(x - 1, y);
                    shots++;
                }
                if (y + 1 < height and image.getPixel(x, y + 1).isDefined) {
                    newPixel += image.getPixel(x, y + 1);
                    shots++;
                }
                if (y - 1 >= 0 and image.getPixel(x, y - 1).isDefined) {
                    newPixel += image.getPixel(x, y - 1);
                    shots++;
                }
                if (shots != 0) {
                    newPixel.red /= shots;
                    newPixel.green /= shots;
                    newPixel.blue /= shots;
                    newPixel.isDefined = true;
                }
                interpolaredImage.setPixel(x, y, newPixel);
                undefinedMap.setPixel(x, y, {255, 255, 255});
            }
        }
    }

    undefinedMap.writeImage();
    return interpolaredImage;
}

int distanceToMarkedLine(const ImageCPU &image) {
    int length{}, shots{};

    for (int i = 0; i < image.getHeight(); i++) {
        for (int j = 0; j < image.getWidth(); j++) {
            if (image.getPixel(j, i) == markerCPU) {
                length += i;
                shots++;
            }
        }
    }

    return length / shots;
}

ImageCPU centralizeLine(const ImageCPU &image, int offset) {
    ImageCPU centralizedImage{"../outImages/centralized.png"};
    auto height = image.getHeight(), width = image.getWidth();
    centralizedImage.setProperties(height, width, image.getChannels());

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            if (y + offset > 0 and y + offset < height) {
                Pixel px = image.getPixel(x, y);
                centralizedImage.setPixel(x, y + offset, px);
                if (px.isDefined) {
                    centralizedImage.definePixel(x, y + offset);
                }
            }
        }
    }

    centralizedImage.writeImage();
    return centralizedImage;
}

void runCpu(std::string imagePath) {
    using namespace std;

    cout << "CPU running" << endl;
    ImageCPU image{std::move(imagePath)};
    image.readImage();
    cout << "ImageCPU height: " << image.getHeight() << endl << "ImageCPU width: " << image.getWidth() << endl;

    // Get angle to rotate image
    const auto houghtResult = houghTransform(image);
    cout << endl << "Hough result: " << houghtResult << endl;
    const auto rotationAngle = houghtResult > 0 ? 90 - houghtResult : -(90 + houghtResult);
    cout << "Rotation angle: " << rotationAngle << endl;

    // Rotate image
    auto outImage = rotateImage(image, rotationAngle);

    // Get shift distance to centralize line
    auto shiftDistance = outImage.getHeight() / 2 - distanceToMarkedLine(outImage);
    cout << "Shift distance: " << shiftDistance << endl;

    // Centralize line
    outImage = centralizeLine(outImage, shiftDistance);

    // Interpolate the image
    outImage = interpolarImage(outImage);
    outImage.setImageName("../outImages/outImage.png");
    outImage.writeImage();
}