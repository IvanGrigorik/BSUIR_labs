#include "hip/hip_runtime.h"
#include "cpu.cuh"
#include <iostream>
#include <utility>

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 255, 0};

/// Returns line angle relative to the horizon (see explanation ahead the function)
/*
 * |                     ----
 * |                 ----
 * |             ---- A (equivalent)
 * | ****************************** horizon line
 * |       A ----
 * |     ----
 * | ----
 * |||||||||||||||||||||||||||||||| image x-axis
 * Return angle A
 */
double houghTransform(const Image &image) {
    Image monochromaticImage{"../outImages/monochromatic.png"};
    monochromaticImage.setProperties(image.getHeight(), image.getWidth(), image.getChannels());
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) {
        houghAccum[i].resize(thetasCount);
    }

    const float step = 180.f / static_cast<float>(thetasCount);

    for (int y = 0; y < image.getHeight(); y++) {
        for (int x = 0; x < image.getWidth(); x++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) {
                continue;
            }
            monochromaticImage.setPixel(x, y, {255, 255, 255});

            double ang = -90;
            for (int h = 0; h < thetasCount; ang += step, h++) {
                int idx = static_cast<int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
                houghAccum[idx][h]++;
            }
        }
    }

    monochromaticImage.writeImage();
    int idx{}, max{};
    for (auto &i: houghAccum) {
        for (int j = 0; j < i.size(); j++) {
            if (i[j] > max) {
                max = i[j];
                idx = j;
            }
        }
    }

    return -90 + step * static_cast<float>(idx);
}

Image rotateImage(const Image &image, const double angle) {
    Image rotatedImage{"../outImages/rotatedImage.png"};

    const auto height = image.getHeight(), width = image.getWidth();

    const int centerX = width / 2, centerY = height / 2;
    rotatedImage.setProperties(height, width, image.getChannels());

    const auto radAngle = degToRad(90 + angle);

    const auto k1 = sin(radAngle);
    const auto k2 = cos(radAngle);

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            auto newX = static_cast<int>(k1 * (x - centerX) + k2 * (y - centerY) + centerX);
            auto newY = static_cast<int>(k1 * (y - centerY) - k2 * (x - centerX) + centerY);

            if (newX < 0 or newX >= width or newY < 0 or newY >= height) {
                continue;
            }

            rotatedImage.setPixel(newX, newY, image.getPixel(x, y));
            rotatedImage.definePixel(newX, newY);
        }
    }

    rotatedImage.writeImage();
    return rotatedImage;
}

Image interpolarImage(const Image &image) {
    Image interpolaredImage{image};
    auto height = image.getHeight(), width = image.getWidth();
    Image undefinedMap{"../outImages/undefinedPixels.png"};
    undefinedMap.setProperties(height, width, image.getChannels());

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            if (!interpolaredImage.getPixel(x, y).isDefined) {
                Pixel newPixel{};

                // rotated image cannot contain more than two undefined pixels in a row
                if (x + 1 < width and x + 2 < width and !image.getPixel(x + 1, y).isDefined and
                    !image.getPixel(x + 2, y).isDefined) {
                    // if trying to interpolate unnecessary parts of line
                    if (x > width / 2) {
                        break;
                    }
                    continue;
                }

                if (x + 1 < width and interpolaredImage.getPixel(x + 1, y).isDefined) {
                    newPixel = interpolaredImage.getPixel(x + 1, y);
                } else if (x - 1 >= 0 and interpolaredImage.getPixel(x - 1, y).isDefined) {
                    newPixel = interpolaredImage.getPixel(x - 1, y);
                }
                if (y + 1 < height and interpolaredImage.getPixel(x, y + 1).isDefined) {
                    newPixel += interpolaredImage.getPixel(x, y + 1);
                } else if (y - 1 >= 0 and interpolaredImage.getPixel(x, y - 1).isDefined) {
                    newPixel += interpolaredImage.getPixel(x, y - 1);
                }
                newPixel.red /= 2;
                newPixel.green /= 2;
                newPixel.blue /= 2;
                newPixel.isDefined = true;
                interpolaredImage.setPixel(x, y, newPixel);
                undefinedMap.setPixel(x, y, {255, 255, 255});
            }
        }
    }

    undefinedMap.writeImage();
    return interpolaredImage;
}

int distanceToMarkedLine(const Image &image) {
    int length{}, shots{};

    for (int i = 0; i < image.getHeight(); i++) {
        for (int j = 0; j < 5; j++) {
            if (image.getPixel(j, i) == marker) {
                length += i;
                shots++;
            }
        }
    }

    return length / shots;
}

Image centralizeLine(const Image &image, int offset) {
    Image centralizedImage{"../outImages/centralized.png"};
    auto height = image.getHeight(), width = image.getWidth();
    centralizedImage.setProperties(height, width, image.getChannels());

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            if (y + offset > 0 and y + offset < height) {
                Pixel px = image.getPixel(x, y);
                centralizedImage.setPixel(x, y + offset, px);
                if (px.isDefined) {
                    centralizedImage.definePixel(x, y + offset);
                }
            }
        }
    }

    centralizedImage.writeImage();
    return centralizedImage;
}

void runCpu(std::string imagePath) {
    using namespace std;

    cout << "CPU running" << endl;
    Image image{std::move(imagePath)};
    image.readImage();
    cout << "Image height: " << image.getHeight() << endl << "Image width: " << image.getWidth() << endl;

    //     Get angle to rotate image
    const auto houghtResult = houghTransform(image);
    cout << endl << "Hough result: " << houghtResult << endl;
    const auto rotationAngle = houghtResult > 0 ? 90 - houghtResult : -(90 + houghtResult);
    cout << "Rotation angle: " << rotationAngle << endl;

    // Rotate image
    auto outImage = rotateImage(image, rotationAngle);

    // Get shift distance to centralize line
    auto shiftDistance = outImage.getHeight() / 2 - distanceToMarkedLine(outImage);
    cout << "Shift distance: " << shiftDistance << endl;

    // Centralize line
    outImage = centralizeLine(outImage, shiftDistance);

    // Interpolate the image
    outImage = interpolarImage(outImage);
    outImage.setImageName("../outImages/outImage.png");
    outImage.writeImage();
}