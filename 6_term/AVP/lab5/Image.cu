//
// Created by sifi on 4/8/23.
//

#include "Image.cuh"
#include "utilities.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

using namespace std;
#include <iostream>

void ImageCPU::readImage() {
    auto image = stbi_load(imageName.c_str(), &width, &height, &channels, 3);
    if (height == 0) {
        perror("ImageCPU opening failed");
        exit(-1);
    }

    imageMatrix.resize(height);

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            // In format RGB (+0 - R; +1 - G; +2 - B)
            imageMatrix[i].push_back(
                    {image[(i * width + j) * 3], image[(i * width + j) * 3 + 1], image[(i * width + j) * 3 + 2]});
        }
    }
}

void ImageCPU::writeImage() const {
    auto *image = new unsigned char[height * width * channels];

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            image[(i * width + j) * 3] = imageMatrix[i][j].red;
            image[(i * width + j) * 3 + 1] = imageMatrix[i][j].green;
            image[(i * width + j) * 3 + 2] = imageMatrix[i][j].blue;
        }
    }

    stbi_write_png(imageName.c_str(), width, height, channels, image, width * channels);
}

Pixel ImageCPU::getPixel(int x, int y) const {
    return imageMatrix[y][x];
}

void ImageCPU::setPixel(int x, int y, const Pixel px) {
    imageMatrix[y][x] = px;
}

void ImageCPU::definePixel(int x, int y) {
    imageMatrix[y][x].isDefined = true;
}

int ImageCPU::getHeight() const {
    return height;
}

int ImageCPU::getWidth() const {
    return width;
}

int ImageCPU::getChannels() const {
    return channels;
}

void ImageCPU::setProperties(int newHeight, int newWidth, int newChannels) {
    height = newHeight;
    width = newWidth;
    channels = newChannels;

    if (!imageMatrix.empty()) {
        imageMatrix.clear();
    }

    imageMatrix.resize(height);
    for (int i = 0; i < height; i++) {
        imageMatrix[i].resize(width);
    }
}

void ImageCPU::setImageName(const string &name) {
    ImageCPU::imageName = name;
}

bool Pixel::operator==(const Pixel &rhs) const {
    return red == rhs.red && green == rhs.green && blue == rhs.blue;
}

bool Pixel::operator!=(const Pixel &rhs) const {
    return !(rhs == *this);
}

Pixel &Pixel::operator+=(const Pixel &rhs) {
    red += rhs.red;
    green += rhs.green;
    blue += rhs.blue;

    return *this;
}

// ImageGPU implementation
void ImageGPU::readImage() {
    auto image = stbi_load(imagePath.c_str(), &width, &height, &channels, 3);
    if (height == 0) {
        perror("ImageCPU opening failed");
        exit(-1);
    }

    auto imageMatrix = new GPUPixel[height * width];

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            // In format RGB (+0 - R; +1 - G; +2 - B)
            imageMatrix[i * width + j] = {image[(i * width + j) * 3], image[(i * width + j) * 3 + 1],
                                          image[(i * width + j) * 3 + 2], true};
        }
    }

    // Malloc pitched memory for kernels
    CUDA_ASSERT(hipMallocPitch(&devData, &pitch, width * sizeof(GPUPixel), height))
    CUDA_ASSERT(hipMemcpy2D(devData, pitch, imageMatrix, width * sizeof(GPUPixel), width * sizeof(GPUPixel), height,
                             hipMemcpyHostToDevice))
}
void ImageGPU::setProperties(int newHeight, int newWidth, int newChannels) {
    height = newHeight;
    width = newWidth;
    channels = newChannels;

    if (pitch != 0) {
        CUDA_ASSERT(hipFree(&devData));
    }
    auto imageMatrix = new GPUPixel[height * width];
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            // In format RGB (+0 - R; +1 - G; +2 - B), plus "is defined" variable
            imageMatrix[i * width + j] = {0, 0, 0, false};
        }
    }
    CUDA_ASSERT(hipMallocPitch(&devData, &pitch, width * sizeof(GPUPixel), height))
    CUDA_ASSERT(hipMemcpy2D(devData, pitch, imageMatrix, width * sizeof(GPUPixel), width * sizeof(GPUPixel), height,
                             hipMemcpyHostToDevice))
}
void ImageGPU::writeImage() const {
    auto imageMatrix = new GPUPixel[height * width];

    CUDA_ASSERT(hipMemcpy2D(imageMatrix, width * sizeof(GPUPixel),//
                             devData, pitch,                       //
                             width * sizeof(GPUPixel), height,     //
                             hipMemcpyDeviceToHost))

    auto *image = new unsigned char[height * width * channels];

    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            image[(i * width + j) * 3] = imageMatrix[i * width + j].red;
            image[(i * width + j) * 3 + 1] = imageMatrix[i * width + j].green;
            image[(i * width + j) * 3 + 2] = imageMatrix[i * width + j].blue;
        }
    }

    stbi_write_png(imagePath.c_str(), width, height, channels, image, width * channels);
}
