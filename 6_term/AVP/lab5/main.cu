#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <tuple>
#include <algorithm>
#include "Image.h"

#define degToRad(val) (val * M_PI / 180)

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 0, 0};

void houghTransform(const Image &image) {
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) {
        houghAccum[i].resize(thetasCount);
    }


    const int step = 180 / thetasCount >= 0 ? 1 : 180 / thetasCount;
    for (int x = 0; x < image.getWidth(); x++) {
        for (int y = 0; y < image.getHeight(); y++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) {
                continue;
            }

            // Create all possible variants of thetas

            // Iterate through all possible lines
            for (int k = -90, h = 0; k < 90; k += step, h++) {
                double idx = maxDist + x * cos(degToRad(k)) + y * sin(degToRad(k));
                houghAccum[static_cast<int>(idx)][h]++;
            }
        }
    }
}

std::tuple<int, int> findMax() {

    
    for (int i = 0; i < houghAccum.size(); i++){

    }

    return std::make_tuple<>(1, 2);
}

int main() {
    using namespace std;
    Image image{"../images/straight_line.png"};
    cout << "Image height: " << image.getHeight() << endl << "Image width: " << image.getWidth() << endl;

    houghTransform(image);

    cout << endl << "Hough result: " << endl;

    for (auto &i: houghAccum) {
        for (auto j: i) {
            cout << j << ' ';
        }
        cout << endl;
    }

    return 0;
}
