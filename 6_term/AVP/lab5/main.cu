#include "hip/hip_runtime.h"
#include "Image.cuh"
#include "cpu.cuh"
#include "utilities.cuh"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/reduce.h>

using namespace std;

// Hough ///////////////////////////////////////////////////////////////////////////////////////////////////////////////

__device__ GPUPixel markerGPU{0, 255, 0};
__device__ GPUPixel thresholdGPU{20, 20, 20};

__global__ void houghKernel(const ImageGPU image, int *houghAccum,//NOLINT (to disable clang-tidy)
                            float step, int thetasCount, int maxDist) {
    const unsigned x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned y = threadIdx.y + blockDim.y * blockIdx.y;

    if (y >= image.height or x >= image.width) {
        return;
    }
    //    if(x == 20)

    auto row = getRow(image.devData, image.pitch, y);
    auto element = row[x];
    //    printf("x:%d y:%d: %d:%d:%d\n", x, y, element.red, element.green, element.blue);

    if (abs(element.red - markerGPU.red) >= thresholdGPU.red or      //
        abs(element.green - markerGPU.green) >= thresholdGPU.green or//
        abs(element.blue - markerGPU.blue) >= thresholdGPU.blue) {
        return;
    }

    double ang = -90;
    for (int h = 0; h < thetasCount; ang += step, h++) {
        int idx = static_cast<int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
        atomicAdd(&houghAccum[thetasCount * idx + h], 1);
        //        if (idx == 44 and h == 60) {
        //            printf("X:Y: %d:%d\n", x, y);
        //        }
    }
}

double houghGPU(const ImageGPU &image, const dim3 numBlocks, const dim3 threadsPerBlock) {
    int *houghAccum;
    const int maxDist = static_cast<int>(round(sqrt(pow(image.height, 2) + pow(image.width, 2))));
    const int thetasCount = 2 * (image.height + image.width) - 4;

    auto accumLen = 2 * maxDist * thetasCount;
    CUDA_ASSERT(hipMallocManaged(&houghAccum, accumLen * sizeof(int)))
    CUDA_ASSERT(hipMemset(houghAccum, 0, accumLen * sizeof(int)))

    const float step = 180.f / static_cast<float>(thetasCount);

    houghKernel<<<numBlocks, threadsPerBlock>>>(image, houghAccum, step, thetasCount, maxDist);
    CUDA_ASSERT(hipDeviceSynchronize())

    int idx{}, max{};
    for (int i = 0; i < maxDist * 2; i++) {
        for (int j = 0; j < thetasCount; j++) {
            if (houghAccum[i * thetasCount + j] > max) {
                max = houghAccum[i * thetasCount + j];
                idx = j;
            }
        }
    }

    auto angle = -90 + step * static_cast<float>(idx);
    return angle;
}


// Rotation ///////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void rotateKernel(const ImageGPU sourceImage, ImageGPU destinationImage,//NOLINT (to disable clang-tidy)
                             const double k1, const double k2, const int centerX, const int centerY) {
    // Cast, 'cause need sign to further calculations
    const int x = static_cast<int>(threadIdx.x + blockDim.x * blockIdx.x);
    const int y = static_cast<int>(threadIdx.y + blockDim.y * blockIdx.y);

    if (y >= sourceImage.height or x >= sourceImage.width or y >= destinationImage.height or
        x >= destinationImage.width) {
        return;
    }

    int newX = static_cast<int>(k1 * (x - centerX) + k2 * (y - centerY) + centerX);
    int newY = static_cast<int>(k1 * (y - centerY) - k2 * (x - centerX) + centerY);

    if (newX < 0 or newX >= sourceImage.width or newY < 0 or newY >= sourceImage.height) {
        return;
    }


    auto srcRow = getRow(sourceImage.devData, sourceImage.pitch, y);
    auto sourceElement = srcRow[x];

    auto dstRow = getRow(destinationImage.devData, destinationImage.pitch, newY);
    dstRow[newX] = sourceElement;
}

ImageGPU rotateImageGPU(const ImageGPU &image, const dim3 numBlocks, const dim3 threadsPerBlock,
                        const double rotationAngle) {
    ImageGPU rotatedImage{"../rotatedImage.png"};
    const int centerX = image.width / 2, centerY = image.height / 2;

    rotatedImage.setProperties(image.height, image.width, image.channels);
    const auto radAngle = degToRad(90 + rotationAngle);

    const auto k1 = sin(radAngle);
    const auto k2 = cos(radAngle);

    rotateKernel<<<numBlocks, threadsPerBlock>>>(image, rotatedImage, k1, k2, centerX, centerY);
    CUDA_ASSERT(hipDeviceSynchronize());

    return rotatedImage;
}

// Centralize line ////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void distanceToMarkedKernel(const ImageGPU image, unsigned *accum, unsigned *shotVal) {//NOLINT
    const int x = static_cast<int>(threadIdx.x + blockDim.x * blockIdx.x);
    const int y = static_cast<int>(threadIdx.y + blockDim.y * blockIdx.y);

    if (y >= image.height or x >= image.width) {
        return;
    }

    auto row = getRow(image.devData, image.pitch, y);
    auto element = row[x];
    if (element.isDefined and               //
        element.red == markerGPU.red and    //
        element.green == markerGPU.green and//
        element.blue == markerGPU.blue) {
        atomicAdd(accum, y);
        atomicAdd(shotVal, 1);
    }
}

__global__ void centralizeKernel(const ImageGPU sourceImage, const ImageGPU destinationImage, int offset) {//NOLINT
    const int x = static_cast<int>(threadIdx.x + blockDim.x * blockIdx.x);
    const int y = static_cast<int>(threadIdx.y + blockDim.y * blockIdx.y);

    if (y >= sourceImage.height or x >= sourceImage.width or y >= destinationImage.height or
        x >= destinationImage.width) {
        return;
    }

    auto srcRow = getRow(sourceImage.devData, sourceImage.pitch, y);
    auto dstRow = getRow(destinationImage.devData, destinationImage.pitch, y + offset);

    if (y + offset > 0 and y + offset < destinationImage.height) {
        dstRow[x] = srcRow[x];
    }
}

ImageGPU centralizeLine(const ImageGPU &image, const dim3 numBlocks, const dim3 threadsPerBlock) {
    // Do preparation calculates
    unsigned *heightAccum;
    CUDA_ASSERT(hipMallocManaged(&heightAccum, image.width * sizeof(unsigned)))
    CUDA_ASSERT(hipMemset(heightAccum, 0, 1))
    unsigned *shotValue;
    CUDA_ASSERT(hipMallocManaged(&shotValue, image.width * sizeof(unsigned)))
    CUDA_ASSERT(hipMemset(shotValue, 0, 1))

    distanceToMarkedKernel<<<numBlocks, threadsPerBlock>>>(image, heightAccum, shotValue);
    CUDA_ASSERT(hipDeviceSynchronize())

    auto average = static_cast<int>(*heightAccum / *shotValue);
    auto shiftDistance = image.height / 2 - average;
    cout << "Shift distance: " << shiftDistance << endl;

    ImageGPU centralizedImage{"../centralized.png"};
    centralizedImage.setProperties(image.height, image.width, image.channels);

    centralizeKernel<<<numBlocks, threadsPerBlock>>>(image, centralizedImage, shiftDistance);
    CUDA_ASSERT(hipDeviceSynchronize())
    return centralizedImage;
}

// Interpolate ////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void interpolateKernel(const ImageGPU sourceImage, const ImageGPU destinationImage,//NOLINT
                                  const ImageGPU undefinedMap) {                              //NOLINT
    const int x = static_cast<int>(threadIdx.x + blockDim.x * blockIdx.x);
    const int y = static_cast<int>(threadIdx.y + blockDim.y * blockIdx.y);

    // Only one image can be compared, because all images have similar sizes
    if (y >= destinationImage.height or x >= destinationImage.width) {
        return;
    }

    auto srcRow = getRow(sourceImage.devData, sourceImage.pitch, y);
    if (!srcRow[x].isDefined) {
        return;
    }

    auto dstRow = getRow(destinationImage.devData, destinationImage.pitch, y);
    auto mapRow = getRow(undefinedMap.devData, undefinedMap.pitch, y);

    // rotated image cannot contain more than two undefined pixels in a row, thus we can
    if (x + 2 < sourceImage.width and !srcRow[x + 1].isDefined and !srcRow[x + 2].isDefined) {
        return;
    }

    int r{}, g{}, b{}, shots{};

    // Compare 4 pixels around
    if (x + 1 < sourceImage.width and srcRow[x + 1].isDefined) {
        r += srcRow[x + 1].red;
        g += srcRow[x + 1].green;
        b += srcRow[x + 1].blue;
        shots++;
    } else if (x - 1 >= 0 and srcRow[x - 1].isDefined) {
        r += srcRow[x - 1].red;
        g += srcRow[x - 1].green;
        b += srcRow[x - 1].blue;
        shots++;
    }
    if (y + 1 < sourceImage.height) {
        auto rowP1 = getRow(destinationImage.devData, destinationImage.pitch, y + 1);
        if (rowP1[x].isDefined) {
            r += rowP1[x].red;
            g += rowP1[x].green;
            b += rowP1[x].blue;
            shots++;
        }
    }
    if (y - 1 >= 0) {
        auto rowM1 = getRow(destinationImage.devData, destinationImage.pitch, y - 1);
        if (rowM1[x].isDefined) {
            r += rowM1[x].red;
            g += rowM1[x].green;
            b += rowM1[x].blue;
            shots++;
        }
    }
    dstRow[x] = {static_cast<uint8_t>(r / shots),//
                 static_cast<uint8_t>(g / shots),//
                 static_cast<uint8_t>(b / shots), true};
}

ImageGPU interpolateImage(const ImageGPU &image, const dim3 numBlocks, const dim3 threadsPerBlock) {

    ImageGPU interpolatedImage{"../interpolated.png"};
    interpolatedImage.setProperties(image.height, image.width, image.channels);

    ImageGPU undefinedMap{"../undefinedPixels.png"};
    undefinedMap.setProperties(image.height, image.width, image.channels);

    interpolateKernel<<<numBlocks, threadsPerBlock>>>(image, interpolatedImage, undefinedMap);
    CUDA_ASSERT(hipDeviceSynchronize())
    return interpolatedImage;
}
// Main function //////////////////////////////////////////////////////////////////////////////////////////////////////
void rotateLine(const std::string &imagePath) {
    ImageGPU image{imagePath};
    image.readImage();

    dim3 threadsPerBlock(30, 30);// change 20 to 30
    dim3 numBlocks{(image.width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (image.height + threadsPerBlock.y - 1) / threadsPerBlock.y};

    // Get angle to rotate the image
    const auto houghtResult = houghGPU(image, numBlocks, threadsPerBlock);
    std::cout << "Hough result: " << houghtResult << endl;
    const auto rotationAngle = houghtResult > 0 ? 90 - houghtResult : -(90 + houghtResult);
    cout << "Rotation angle: " << rotationAngle << endl;

    // Rotate the image
    auto rotatedImage = rotateImageGPU(image, numBlocks, threadsPerBlock, rotationAngle);
    rotatedImage.writeImage();

    // Shift the marked strip to the center
    auto centralizedImage = centralizeLine(rotatedImage, numBlocks, threadsPerBlock);
    centralizedImage.writeImage();
    auto interpolated = interpolateImage(centralizedImage, numBlocks, threadsPerBlock);
    interpolated.writeImage();
}

int main() {
    auto imagePath = "../images/man.png";

    cout << "GPU implementation: " << endl;
    rotateLine(imagePath);

    cout << endl << endl;
    cout << "CPU implementation: " << endl;
    runCpu(imagePath);

    return 0;
}
