#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <iomanip>
#include "Image.h"

#define degToRad(val) (val * M_PI / 180)

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 0, 0};

/// Returns line angle relative to the horizon (see explanation ahead the function)
/*
 * |                     ----
 * |                 ----
 * |             ---- A (equivalent)
 * | ****************************** horizon line
 * |       A ----
 * |     ----
 * | ----
 * |||||||||||||||||||||||||||||||| image x-axis
 * Return angle A
 */
int houghTransform(const Image &image) {
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) {
        houghAccum[i].resize(thetasCount);
    }

    const float step = 180.f / static_cast<float>(thetasCount);
    std::vector<float> thetas;


    for (int x = 0; x < image.getHeight(); x++) {
        for (int y = 0; y < image.getWidth(); y++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) {
                continue;
            }

            // Iterate through all possible lines (need to rewrite)
            double ang = -90;
            for (int h = 0; h < thetasCount; ang += step, h++) {
                int idx = static_cast <int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
                houghAccum[idx][h]++;
            }
        }
    }

    ulong idx{}, max{};
    for (auto &i: houghAccum) {
        for (int j = 0; j < i.size(); j++) {
            if (i[j] > max) {
                max = i[j];
                idx = j;
            }
        }
    }

    double angle{-90};
    for (int i = 0; i < idx; i++) {
        angle += step;
    }
    return static_cast<int>(angle);
}


int main() {
    using namespace std;
    Image image{"../images/line65.png"};
    cout << "Image height: " << image.getHeight() << endl << "Image width: " << image.getWidth() << endl;

    //
    auto angle = houghTransform(image);



    cout << endl << "Hough result: " << angle << endl;

    return 0;
}
