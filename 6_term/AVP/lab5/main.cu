#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <iomanip>
#include "Image.h"

#define degToRad(val) (val * M_PI / 180)

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 0, 0};

void houghTransform(const Image &image) {
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) {
        houghAccum[i].resize(thetasCount);
    }

    const float step = 180.f / static_cast<float>(thetasCount);
    std::vector<float> thetas;
    thetas.resize(thetasCount);

    double ang = -90;
    while (ang < 90) {
        ang += step;
        thetas.push_back(degToRad(ang));
    }

    for (int x = 0; x < image.getWidth(); x++) {
        for (int y = 0; y < image.getHeight(); y++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) {
                continue;
            }



            // Iterate through all possible lines (need to rewrite)
            for (double k = -90, h = 0; k < 90; k += step, h++) {
                double idx = x * cos(degToRad(k)) + y * sin(degToRad(k));
                houghAccum[static_cast<int>(idx) + maxDist][static_cast<int>(h)]++;
            }
        }
    }


}


int main() {
    using namespace std;
    Image image{"../images/line3x3.png"};
    cout << "Image height: " << image.getHeight() << endl << "Image width: " << image.getWidth() << endl;

    houghTransform(image);

    cout << endl << "Hough result: " << endl;

    for (auto &i: houghAccum) {
        for (auto j: i) {
            cout << setw(2) << j << ' ';
        }
        cout << endl;
    }

    return 0;
}
