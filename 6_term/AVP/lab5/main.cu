#include "hip/hip_runtime.h"
#include "cpu.cuh"
#include "Image.cuh"
#include "utilities.cuh"
#include <iostream>

__device__ GPUPixel markerGPU{0, 255, 0};
__device__ GPUPixel threshold{20, 20, 20};

__global__ void kernel(const ImageGPU image, int *houghAccum,   //NOLINT (to disable clang-tidy)
                       float step, int thetasCount, int maxDist,//
                       int height, int width, size_t pitch) {
    const unsigned x = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned y = threadIdx.y + blockDim.y * blockIdx.y;

    if (y >= height || x >= width) {
        return;
    }

    auto row = reinterpret_cast<GPUPixel *>(reinterpret_cast<uint8_t *>(image.devData) + pitch * y);
    auto element = row[x];
    //    printf("x:%d y:%d: %d:%d:%d\n", x, y, element.red, element.green, element.blue);

    if (element.red - markerGPU.red >= threshold.red or      //
        element.green - markerGPU.green >= threshold.green or//
        element.blue - markerGPU.blue >= threshold.blue) {
        return;
    }

    double ang = -90;
    for (int h = 0; h < thetasCount; ang += step, h++) {
        int idx = static_cast<int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
        atomicAdd(&houghAccum[idx + h], 1);
    }
}


double houghGPU(const ImageGPU &image, const int numBlocks, dim3 threadsPerBlock) {
    using namespace std;
    int *houghAccum;
    const int maxDist = static_cast<int>(round(sqrt(pow(image.height, 2) + pow(image.width, 2))));
    const int thetasCount = 2 * (image.height + image.width) - 4;

    // May be wrong calculate houghAccum len (paranoid, all ok)?
    auto accumLen = 2 * maxDist * thetasCount;
    CUDA_ASSERT(hipMallocManaged(&houghAccum, accumLen * sizeof(int)))
    CUDA_ASSERT(hipMemset(houghAccum, 0, accumLen * sizeof(int)))

    const float step = 180.f / static_cast<float>(thetasCount);

    kernel<<<numBlocks, threadsPerBlock>>>(image, houghAccum, step, thetasCount, maxDist,//
                                           image.height, image.width, image.pitch);
    CUDA_ASSERT(hipDeviceSynchronize())

    int idx{}, max{};
    for (int i = 0; i < maxDist * 2; i++) {
        for (int j = 0; j < thetasCount; j++) {
            if (houghAccum[i * thetasCount + j] > max) {
                max = houghAccum[i * thetasCount + j];
                idx = j;
            }
        }
    }

    auto angle = -90 + step * static_cast<float>(idx);
    cout << angle << endl;

    return angle;
}

void rotateLine(const std::string &imagePath, const int numBlocks, dim3 threadsPerBlock) {
    ImageGPU image{"../images/rgb.png"};
    image.readImage();

    const auto houghtResult = houghGPU(image, numBlocks, threadsPerBlock);
}

int main() {
    using namespace std;

    const int numBlocks = 2;
    dim3 threadsPerBlock(10, 10);

    rotateLine("../images/untitled.png", numBlocks, threadsPerBlock);
    cout << "CPU implementation: " << endl;
    runCpu("../images/untitled.png");

    return 0;
}
