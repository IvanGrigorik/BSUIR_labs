#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <algorithm>
#include <iomanip>
#include "Image.h"

#define degToRad(val) (val * M_PI / 180)
#define radToDeg(val) (val * 180 / M_PI)

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 0, 0};

int houghTransform(const Image &image) {
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) {
        houghAccum[i].resize(thetasCount);
    }

    const float step = 180.f / static_cast<float>(thetasCount);
    std::vector<float> thetas;

    double angelBotBorder = -90;
    while (angelBotBorder < 90) {
        angelBotBorder += step;
        thetas.push_back(degToRad(angelBotBorder));
    }

    for (int x = 0; x < image.getWidth(); x++) {
        for (int y = 0; y < image.getHeight(); y++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) {
                continue;
            }

            // Iterate through all possible lines (need to rewrite)
            for (int h = 0; h < thetasCount; h++) {
                double idx = maxDist + x * cos(thetas[h]) + y * sin(thetas[h]);
                houghAccum[static_cast<int>(idx)][static_cast<int>(h)]++;
            }
        }
    }

    ulong idx{}, max{};
    for (int i = 0; i < houghAccum.size(); i++) {
        for (int j = 0; j < houghAccum[i].size(); j++) {
            if (houghAccum[i][j] > max) {
                max = houghAccum[i][j];
                idx = i * houghAccum.size() + j;
            }
        }
    }

    auto angle = thetas[static_cast<int>(idx % thetasCount)];
    return radToDeg(angle);
}


int main() {
    using namespace std;
    Image image{"../images/line65.png"};
    cout << "Image height: " << image.getHeight() << endl << "Image width: " << image.getWidth() << endl;

    auto angle = houghTransform(image);

    cout << endl << "Hough result: " << angle << endl;

//    for (auto &i: houghAccum) {
//        for (auto j: i) {
//            cout << setw(2) << j << ' ';
//        }
//        cout << endl;
//    }

    return 0;
}
