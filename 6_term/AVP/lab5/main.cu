#include "hip/hip_runtime.h"
#include "Image.cuh"
#include <algorithm>
#include <cmath>
#include <iomanip>
#include <iostream>

#define degToRad(val) (val * M_PI / 180)

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 0, 0};

/// Returns line angle relative to the horizon (see explanation ahead the function)
/*
 * |                     ----
 * |                 ----
 * |             ---- A (equivalent)
 * | ****************************** horizon line
 * |       A ----
 * |     ----
 * | ----
 * |||||||||||||||||||||||||||||||| image x-axis
 * Return angle A
 */
int houghTransform(const Image &image) {
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) { houghAccum[i].resize(thetasCount); }

    const float step = 180.f / static_cast<float>(thetasCount);

    for (int x = 0; x < image.getHeight(); x++) {
        for (int y = 0; y < image.getWidth(); y++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) { continue; }

            double ang = -90;
            for (int h = 0; h < thetasCount; ang += step, h++) {
                int idx = static_cast<int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
                houghAccum[idx][h]++;
            }
        }
    }

    int idx{}, max{};
    for (auto &i: houghAccum) {
        for (int j = 0; j < i.size(); j++) {
            if (i[j] > max) {
                max = i[j];
                idx = j;
            }
        }
    }

    return static_cast<int>(-90 + step * static_cast<float>(idx));
}

Image rotateImage(const Image &image, const int angle) {
    Image newImage{"../outImage.png"};
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    auto centerX = image.getWidth() / 2, centerY = image.getHeight() / 2;
    newImage.setProperties(maxDist, maxDist, image.getChannels());

    auto angCos = cos(degToRad(angle)), angSin = sin(degToRad(angle));
    for (int i = 0; i < image.getHeight(); i++) {
        for (int j = 0; j < image.getWidth(); j++) {
            auto x = static_cast<int>((i - centerX) * angCos + (j - centerY) * angSin);
            auto y = static_cast<int>((j - centerY) * angCos - (i - centerX) * angSin);

//            if (x > maxDist or x < 0 or y > maxDist or y < 0) { continue; }

            newImage.setPixel(x, y, image.getPixel(i, j));
        }
    }

    newImage.writeImage();
}

int main() {
    using namespace std;
    Image image{"../images/straight_line.png"};
    image.readImage();
    cout << "Image height: " << image.getHeight() << endl << "Image width: " << image.getWidth() << endl;

    // Get angle to rotate image
    const auto rotationAngle = houghTransform(image);
    cout << endl << "Hough result: " << rotationAngle << endl;


    Image rotatedImage = rotateImage(image, rotationAngle);

    return 0;
}
