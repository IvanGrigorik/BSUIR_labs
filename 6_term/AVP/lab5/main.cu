#include "hip/hip_runtime.h"
#include "Image.h"
#include <algorithm>
#include <cmath>
#include <iostream>

#define degToRad(val) (val * M_PI / 180)

std::vector<std::vector<int>> houghAccum;
Pixel marker{0, 0, 0};

/// Returns line angle relative to the horizon (see explanation ahead the function)
/*
 * |                     ----
 * |                 ----
 * |             ---- A (equivalent)
 * | ****************************** horizon line
 * |       A ----
 * |     ----
 * | ----
 * |||||||||||||||||||||||||||||||| image x-axis
 * Return angle A
 */
int houghTransform(const Image &image) {
    const int maxDist = static_cast<int>(round(sqrt(pow(image.getHeight(), 2) + pow(image.getWidth(), 2))));
    // Little theta and step optimization
    const int thetasCount = 2 * (image.getHeight() + image.getWidth()) - 4;

    // Memory allocation
    houghAccum.resize(2 * maxDist);
    for (int i = 0; i < 2 * maxDist; i++) { houghAccum[i].resize(thetasCount); }

    const float step = 180.f / static_cast<float>(thetasCount);

    for (int x = 0; x < image.getHeight(); x++) {
        for (int y = 0; y < image.getWidth(); y++) {
            // Find pixel fits the marker
            if (image.getPixel(x, y) != marker) { continue; }

            double ang = -90;
            for (int h = 0; h < thetasCount; ang += step, h++) {
                int idx = static_cast<int>(maxDist + x * cos(degToRad(ang)) + y * sin(degToRad(ang)));
                houghAccum[idx][h]++;
            }
        }
    }

    int idx{}, max{};
    for (auto &i: houghAccum) {
        for (int j = 0; j < i.size(); j++) {
            if (i[j] > max) {
                max = i[j];
                idx = j;
            }
        }
    }

    return static_cast<int>(-90 + step * static_cast<float>(idx));
}


//Image rotateImage(const Image &image, const int angle) {
//
//    Image returnedImage{};
//    for (int i = 0; i < image.getHeight(); i++) {
//        for (int j = 0; j < image.getWidth(); j++) {
//
//        }
//    }
//}

int main() {
    using namespace std;
    Image image{"../images/line65.png"};
    cout << "Image height: " << image.getHeight() << endl
         << "Image width: " << image.getWidth() << endl;


    // Get angle to rotate image
    const auto rotationAngle = houghTransform(image);
    cout << endl
         << "Hough result: " << rotationAngle << endl;

    //    Image rotatedImage = rotateImage(image, rotationAngle);

    return 0;
}
